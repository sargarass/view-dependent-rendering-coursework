#include "hip/hip_runtime.h"
#include "ViewDependentRender.h"
#include "LibResources.h"
#include <hipcub/hipcub.hpp>
#include "SystemManager.h"

#define DEGREE 4
#define NUM_COMP 4
#define NUM_POINTS 16
#define HELP_ARRAYS_COUNT 6

#define NUM_THREADS_MVP 1024
#define NUM_PATCHES_MVP ((NUM_THREADS_MVP) / 16)

#define NUM_THREADS_SPLIT 128
#define NUM_PATCHES_SPLIT ((NUM_THREADS_SPLIT) / 16)

#define NUM_THREADS_ORACLE 192
#define NUM_PATCHES_ORACLE ((NUM_THREADS_ORACLE) / 16)

#define NUM_THREADS_WITHOUT_SHARED 1024
#define NUM_PATCHES_WITHOUT_SHARED ((NUM_THREADS_WITHOUT_SHARED) / 16)

#define NUM_THREADS_WITH_SHARED 256
#define NUM_PATCHES_WITH_SHARED ((NUM_THREADS_WITH_SHARED) / 16)

#define SQR(x) ((x) * (x))
#define NUM_TRIANGLES_IN_PATCH (sizeof(Triangles) / sizeof(glm::vec4) / 3)
#define NUM_POINTS_IN_TRIANGLES (NUM_TRIANGLES_IN_PATCH * 3)

#define X 0
#define Y 1
#define Z 2
#define W 3

#define GET_COMP(patch, point, comp) patch[(comp) * NUM_POINTS + (point)]

__constant__ float gSLU[4][4];
__constant__ float gSRU[4][4];
__constant__ float gSLV[4][4];
__constant__ float gSRV[4][4];

__constant__ float gMVP[4][4];
__constant__ float gIMVP[4][4];
__constant__ float gClipSpacePlanes[6][4];
__constant__ float gWidth;
__constant__ float gHeight;

__constant__ uint8_t triangleThreadPointChooser[NUM_POINTS_IN_TRIANGLES];
__constant__ uint8_t triangleThreadPointChooser2[NUM_POINTS_IN_TRIANGLES];
__constant__ uint8_t triangleThreadPointChooser3[NUM_POINTS_IN_TRIANGLES];

__constant__ uint8_t edgeThreadPointChooser[((DEGREE - 2) * 4) * 3];

#pragma pack(push, 1)
struct PatchPointer{
    union {
        struct {
            float *x;
            float *y;
            float *z;
            float *w;
        };
        float *p[4];
    };
};
#pragma pack(pop)


static __device__ __inline__
float cuda_min(float a, float b) {
    return (a < b)? a : b;
}

static __device__ __inline__
float cuda_max(float a, float b) {
    return (a < b)? b : a;
}

static __device__ __inline__
void warpReduceMin(volatile float *memory) {
    int idx = threadIdx.y * 4 + threadIdx.x;
    if (idx < 8) {
        memory[idx] = cuda_min(memory[idx], memory[idx + 8]);
        memory[idx] = cuda_min(memory[idx], memory[idx + 4]);
        memory[idx] = cuda_min(memory[idx], memory[idx + 2]);
        memory[idx] = cuda_min(memory[idx], memory[idx + 1]);
    }
}

static __device__ __inline__
void warpReduceMax(volatile float *memory) {
    int idx = threadIdx.y * 4 + threadIdx.x;
    if (idx < 8) {
        memory[idx] = cuda_max(memory[idx], memory[idx + 8]);
        memory[idx] = cuda_max(memory[idx], memory[idx + 4]);
        memory[idx] = cuda_max(memory[idx], memory[idx + 2]);
        memory[idx] = cuda_max(memory[idx], memory[idx + 1]);
    }
}

static __device__ __inline__
void warpReduceOr(volatile uint32_t *memory) {
    int idx = threadIdx.y * 4 + threadIdx.x;
    if (idx < 8) {
        memory[idx] |= memory[idx + 8];
        memory[idx] |= memory[idx + 4];
        memory[idx] |= memory[idx + 2];
        memory[idx] |= memory[idx + 1];
    }
}

static __device__ __inline__
void warpReduceAnd(volatile uint32_t *memory) {
    int idx = threadIdx.y * 4 + threadIdx.x;
    if (idx < 8) {
        memory[idx] &= memory[idx + 8];
        memory[idx] &= memory[idx + 4];
        memory[idx] &= memory[idx + 2];
        memory[idx] &= memory[idx + 1];
    }
}


static __device__ __inline__
float clamp(float v, float min, float max) {
    return fmaxf(fminf(v, max), min);
}

static __device__ __inline__
float linearInterpolation(float const &a, float const &b, float const t) {
    return (1.0f - t) * a + t * b;
}

static __device__ __inline__
glm::vec2 linearInterpolation2D(glm::vec2 const &a, glm::vec2 const b, float const t) {
    glm::vec2 res;
    res.x = (1.0 - t) * a.x + t * b.x;
    res.y = (1.0 - t) * a.y + t * b.y;
    return res;
}

static __device__ __inline__
glm::vec4 linearInterpolation4D(glm::vec4 const &a, glm::vec4 const &b, float const &t) {
    glm::vec4 res;
    res.x = (1.0 - t) * a.x + t * b.x;
    res.y = (1.0 - t) * a.y + t * b.y;
    res.z = (1.0 - t) * a.z + t * b.z;
    res.w = (1.0 - t) * a.w + t * b.w;
    return res;
}


static __device__
glm::vec4 bilinearInterpolation4D(float const u, float const v, glm::vec4 const &a, glm::vec4 const &b, glm::vec4 const &c, glm::vec4 const &d) {
    glm::vec4 tmp = linearInterpolation4D(a, b, u);
    glm::vec4 tmp2 = linearInterpolation4D(c, d, u);
    return linearInterpolation4D(tmp, tmp2, v);
}

void VDRender::init(uint64_t gpuMemorySize) {
    /*if (gpuMemorySize < 128ull * 1024ull * 1024ull) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "init", "memory size < 128M invalid");
        exit(-1);
        return;
    }*/

    if (gpuAlignSize * 10 > gpuMemorySize) {
        return;
    }

    m_settings.faceMode = VDFrontFace::NONE;
    m_settings.fillMode = VDFill::FILL;

    GpuStackAllocator& allocator = SystemManager::getInstance()->gpuStackAllocator;
    uint64_t cudaObjects = (5 * sizeof(uint64_t) + sizeof(DecisionBits) + 2 * sizeof(float) * NUM_COMP * NUM_POINTS);
    uint64_t openGLObjects = sizeof(Triangles);
    m_settings.maxQueueSize = (gpuMemorySize - gpuAlignSize * 10) / (cudaObjects + openGLObjects);
    allocator.resize(cudaObjects * m_settings.maxQueueSize);
    m_statistics.maxMemoryQueueSizeInMB = CONVERT_BYTES_TO_MB(m_settings.maxQueueSize * cudaObjects);
    m_statistics.maxMemoryGLBufferSizeInMB = CONVERT_BYTES_TO_MB(m_settings.maxQueueSize * openGLObjects);
    m_glPart.objectsInVBO = 0;

    glGenVertexArrays(1, &m_glPart.vao);
    if (m_glPart.vao == 0) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "init", "glGenVertexArrays == NULL at %s", SourcePos());
        exit(-1);
    }

    glGenBuffers(1, &m_glPart.vbo);
    if (m_glPart.vbo == 0) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "init", "glGenBuffers == NULL at %s", SourcePos());
        exit(-1);
    }

    glGenQueries(1, &m_glPart.query);
    if (m_glPart.query == 0) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "init", "glGenBuffers == NULL at %s", SourcePos());
        exit(-1);
    }
    glNamedBufferDataEXT(m_glPart.vbo, m_settings.maxQueueSize * openGLObjects, NULL, GL_DYNAMIC_DRAW);
    m_glPart.buffer.init(m_glPart.vbo);
    printOpenGLError();
    if (!m_glPart.shader.load("../share/shaders/BezierPatchVertexShader.glsl",
                    "../share/shaders/BezierPatchFragmentShader.glsl")) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "init",
                                 "shader was not load");
        exit(-1);
    }


    float SLU[4][4] = {{1.0f, 1.0f/2.0f, 1.0f/4.0f, 1.0f / 8.0f},
                      {0.0f, 1.0f/2.0f, 1.0f/2.0f, 3.0f / 8.0f},
                      {0.0f, 0.0f,      1.0f/4.0f, 3.0f / 8.0f},
                      {0.0f, 0.0f,      0.0f,      1.0f / 8.0f}};

    float SRU[4][4] = {{1.0f / 8.0f, 0.0f,      0.0f,        0.0f},
                      {3.0f / 8.0f, 1.0f/4.0f, 0.0f,        0.0f},
                      {3.0f / 8.0f, 1.0f/2.0f, 1.0f / 2.0f, 0.0f},
                      {1.0f / 8.0f, 1.0f/4.0f, 1.0f / 2.0f, 1.0f}};

    float SLV[4][4] = {{1.0f,     0.0f,          0,          0    },
                      {1.0f/2.0f, 1.0f/2.0f,     0,          0     },
                      {1.0f/4.0f, 1.0/2.0f,  1.0f/4.0f,     0.0f   },
                      {1.0f/8.0f, 3.0f/8.0f, 3.0f/8.0f, 1.0f / 8.0f}};

    float SRV[4][4] = {{1.0f/8.0f, 3.0f/8.0f, 3.0f/8.0f, 1.0/8.0f},
                       {0.0f, 1.0f/4.0f, 1.0f/2.0f, 1.0f/4.0f},
                      {0.0f, 0.0f, 1.0/2.0f, 1.0f/2.0f},
                      {0.0f, 0.0f, 0.0f, 1.0f}};

    uint8_t trianglePointChooser[18 * 3] = /*{0, 4, 1, 5, 2, 6, 4, 8, 5, 9, 6, 10, 8, 12, 9, 13, 10, 14,
                                   1, 1, 2, 2, 3, 3, 5, 5, 6, 6, 7, 7, 9, 9, 10, 10, 11, 11,
                                   4, 5, 5, 6, 6, 7, 8, 9, 9, 10, 10, 11, 12, 13, 13, 14, 14, 15}*/
                                   {0, 1, 4, 4, 1, 5, 1, 2, 5, 5, 2, 6, 2, 3, 6, 6, 3, 7, 4, 5, 8,
                                   8, 5, 9, 5, 6, 9, 9, 6, 10, 6, 7, 10, 10, 7, 11, 8, 9, 12, 12,
                                   9, 13, 9, 10, 13, 13, 10, 14, 10, 11, 14, 14, 11, 15};
    uint8_t trianglePointChooser2[18 * 3] = {
        1, 0, 0, 1, 4, 4, 2, 1, 1, 2, 5, 5, 3, 2, 2, 3, 6, 6, 5,
        4, 4, 5, 8, 8, 6, 5, 5, 6, 9, 9, 7, 6, 6, 7, 10, 10, 9,
        8, 8, 9, 12, 12, 10, 9, 9, 10, 13, 13, 11, 10, 10, 11, 14, 14
    };

    uint8_t trianglePointChooser3[18 * 3] = {
        4, 4, 1, 5, 5, 1, 5, 5, 2, 6, 6, 2, 6, 6, 3, 7, 7, 3, 8,
        8, 5, 9, 9, 5, 9, 9, 6, 10, 10, 6, 10, 10, 7, 11, 11, 7, 12,
        12, 9, 13, 13, 9, 13, 13, 10, 14, 14, 10, 14, 14, 11, 15, 15, 11
    };

    uint8_t edgePointChooser[8 * 3] = {1, 2, 4,  8,  7,  11, 13, 14,
                                       0, 0, 0,  0,  3,  3,  12, 12,
                                       3, 3, 12, 12, 15, 15, 15, 15};

    hipMemcpyToSymbol(HIP_SYMBOL(gSLU), SLU, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(gSRU), SRU, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(gSLV), SLV, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(gSRV), SRV, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(triangleThreadPointChooser), trianglePointChooser, sizeof(uint8_t) * 18 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(triangleThreadPointChooser2), trianglePointChooser2, sizeof(uint8_t) * 18 * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(triangleThreadPointChooser3), trianglePointChooser3, sizeof(uint8_t) * 18 * 3);

    hipMemcpyToSymbol(HIP_SYMBOL(edgeThreadPointChooser), edgePointChooser, sizeof(uint8_t) * 8 * 3);
    glm::vec4 clipSpacePlanes[6] = {{1, 0, 0, 1.0000001},{-1, 0, 0, 1.0000001},{0, 1, 0, 1.0000001}, {0, -1, 0, 1.0000001}, {0, 0, 1, 1.0000001}, {0, 0, -1, 1.0000001}};
    hipMemcpyToSymbol(HIP_SYMBOL(gClipSpacePlanes), clipSpacePlanes, sizeof(glm::vec4) * 6);
}

static dim3 gridConfigure(uint64_t problemSize, dim3 block) {
    dim3 MaxGridDim = {(uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[0],
                       (uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[1],
                       (uint)LibResouces::getCudaProperties(0).maxGridDimensionSize[2]};
    dim3 gridDim = {1, 1, 1};

    uint64_t blockSize = block.x * block.y * block.z;
    // По z
    if (problemSize > MaxGridDim.y * MaxGridDim.x * blockSize) {
        gridDim.z = problemSize / MaxGridDim.x * MaxGridDim.y * blockSize;
        problemSize = problemSize % MaxGridDim.x * MaxGridDim.y * blockSize;
    }
    // По y
    if (problemSize > MaxGridDim.x * blockSize) {
        gridDim.y = problemSize / MaxGridDim.x * blockSize;
        problemSize = problemSize % MaxGridDim.x * blockSize;
    }

    gridDim.x = (problemSize + blockSize - 1) / blockSize;
    return gridDim;
}

static dim3 gridConfigureZ(uint64_t problemSize, dim3 block) {
    dim3 block_tmp = block;
    block_tmp.x = block_tmp.y = 1;
    return gridConfigure(problemSize, block_tmp);
}

VDRender::~VDRender() {
    deinit();
}

void VDRender::deinit() {
    if (m_glPart.vbo) {
        glDeleteBuffers(1, &m_glPart.vbo);
        m_glPart.vbo = 0;
    }
    if (m_glPart.vao) {
        glDeleteVertexArrays(1, &m_glPart.vao);
        m_glPart.vao = 0;
    }
    if (m_glPart.query) {
        glDeleteQueries(1, &m_glPart.query);
    }
    m_glPart.buffer.deinit();

    for (auto i = m_models.begin(); i != m_models.end(); i++) {
        if ((*i).second.x) {
            gpuFree((*i).second.x);
            (*i).second.x = 0;
        }
        if ((*i).second.y) {
            gpuFree((*i).second.y);
            (*i).second.y = 0;
        }
        if ((*i).second.z) {
            gpuFree((*i).second.z);
            (*i).second.z = 0;
        }
        if ((*i).second.w) {
            gpuFree((*i).second.w);
            (*i).second.w = 0;
        }
    }
}

void VDRender::beginFrame() {
    m_statistics.clear();
    m_glPart.objectsInVBO = 0;
}

void VDRender::endFrame() {
    flushGL();
}

void VDRender::updateParameters(glm::mat4 const &MVP, uint32_t const &width, uint32_t const &height) {
    glm::mat4 IMVP = glm::inverse(MVP);
    float tMPV[4][4];
    float tIMPV[4][4];
    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            tMPV[j][i] = MVP[i][j];
            tIMPV[j][i] = IMVP[i][j];
        }
    }
    float fwidth = width;
    float fheight = height;
    hipMemcpyToSymbol(HIP_SYMBOL(gMVP), tMPV, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(gIMVP), tIMPV, sizeof(float) * 16);
    hipMemcpyToSymbol(HIP_SYMBOL(gWidth), &fwidth, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(gHeight), &fheight, sizeof(float));
}

void VDRender::setFill(VDFill fillMode) {
    m_settings.fillMode = fillMode;
}

void VDRender::setFrontFace(VDFrontFace face) {
    m_settings.faceMode = face;
}

void VDRender::drawGL(uint64_t size, uint64_t level) {
    GLuint64 elapsed_time;
    glBeginQuery(GL_TIME_ELAPSED, m_glPart.query);

    printOpenGLError();
    uint64_t patches = size;
    size = size * NUM_POINTS_IN_TRIANGLES;
    m_glPart.shader.bind();
    m_glPart.shader.setVal("color", glm::vec4(1.0 / level, 0, 0, 1.0f));
    glBindVertexArray(m_glPart.vao);
    glBindBuffer(GL_ARRAY_BUFFER, m_glPart.vbo);
    glEnableVertexAttribArray(0);
    glVertexAttribPointer(0, NUM_COMP, GL_FLOAT, GL_FALSE, 0, 0);

    if (m_settings.fillMode == VDFill::FILL) {
        glPolygonMode(GL_FRONT_AND_BACK, GL_FILL);
    } else {
        if (m_settings.fillMode == VDFill::LINES) {
            glPolygonMode(GL_FRONT_AND_BACK, GL_LINE);
        }
    }
    glDrawArrays(GL_TRIANGLES, 0, size);

    glBindVertexArray(0);
    m_glPart.shader.unbind();

    glEndQuery(GL_TIME_ELAPSED);

    int done = 0;
    while (!done) {
        glGetQueryObjectiv(m_glPart.query, GL_QUERY_RESULT_AVAILABLE, &done);
    }
    glGetQueryObjectui64v(m_glPart.query, GL_QUERY_RESULT, &elapsed_time);
    printOpenGLError();

    m_statistics.glDrawNanoseconds += elapsed_time;
    m_statistics.patchesCountFinal += patches;
    m_statistics.trianglesCount += NUM_TRIANGLES_IN_PATCH * patches;
    m_statistics.drawCallsCounter++;
}

static __device__ __inline__
uint64_t getGlobalIdx3DZ() {
    uint64_t blockId = blockIdx.x
                 + blockIdx.y * gridDim.x
                 + gridDim.x * gridDim.y * blockIdx.z;
    return blockId * blockDim.z + threadIdx.z;
}

static __device__ __inline__
uint64_t getGlobalIdx3DZXY()
{
    uint64_t blockId = blockIdx.x
             + blockIdx.y * gridDim.x
             + gridDim.x * gridDim.y * blockIdx.z;
    return blockId * (blockDim.x * blockDim.y * blockDim.z)
              + (threadIdx.z * (blockDim.x * blockDim.y))
              + (threadIdx.y * blockDim.x)
              + threadIdx.x;
}

static __device__ __inline__
uint64_t getIdx() {
    return threadIdx.y * DEGREE + threadIdx.x;
}

static __global__
void kernelTransfer(RenderModel dest, BezierPatch const * const src, uint64_t const size) {
    uint64_t patchId = getGlobalIdx3DZ();
    if (patchId >= size) {
        return;
    }
    uint64_t idx = getIdx();
    uint64_t patchOut = patchId * NUM_POINTS + idx;

    dest.x[patchOut] = src[patchId].row[idx].x;
    dest.y[patchOut] = src[patchId].row[idx].y;
    dest.z[patchOut] = src[patchId].row[idx].z;
    dest.w[patchOut] = src[patchId].row[idx].w;
}

void VDRender::runKernelTransfer(RenderModel &dest, BezierPatch* src, uint64_t size) {
    size_t patches = NUM_PATCHES_WITHOUT_SHARED;
    dim3 block = dim3(DEGREE, DEGREE, patches);
    dim3 gridDim = gridConfigureZ(size, block);
    kernelTransfer<<<gridDim, block>>>(dest, src, size);
    cudaCheckErrors("runKernelTransfer");
}

bool VDRender::loadPatches(std::string const modelName, BezierPatch const *ramPatches, uint64_t size) {
    if (this->m_models.find(modelName) != m_models.end()) {
        return false;
    }

    BezierPatch* patch_tmp = gpuMalloc<BezierPatch>(size);
    RenderModel model;
    model.x = gpuMalloc<float>(size * NUM_POINTS);
    model.y = gpuMalloc<float>(size * NUM_POINTS);
    model.z = gpuMalloc<float>(size * NUM_POINTS);
    model.w = gpuMalloc<float>(size * NUM_POINTS);
    model.size = size;

    if (patch_tmp == nullptr || model.w == nullptr || model.x == nullptr || model.y == nullptr || model.z == nullptr) {
        if (patch_tmp) {
            gpuFree(patch_tmp);
        }
        if (model.w) {
            gpuFree(model.w);
        }
        if (model.x) {
            gpuFree(model.x);
        }
        if (model.y) {
            gpuFree(model.y);
        }
        if (model.z) {
            gpuFree(model.z);
        }

        Log::getInstance().write(LOG_MESSAGE_TYPE::WARNING, "ViewDependentRender", "loadPatches", "Model %s was not load!", modelName.c_str());
        return false;
    }
    hipMemcpy(patch_tmp, ramPatches, size * sizeof(BezierPatch), hipMemcpyHostToDevice);
    runKernelTransfer(model, patch_tmp, size);
    m_models.insert(std::pair<std::string, RenderModel>(modelName, std::move(model)));
    gpuFree(patch_tmp);
    return true;
}

static __device__ __inline__
float cudaDot4D(float const * const a, glm::vec4 const &b) {
    return a[0] * b[0] + a[1] * b[1] + a[2] * b[2] + a[3] * b[3];
}

__device__ __inline__
float cudaDot4D(glm::vec4 const &a, float const * const b) {
    return a[0] * b[0] + a[1] * b[1] + a[2] * b[2] + a[3] * b[3];
}

__device__ __inline__
float cudaDot4D(float const * const patch, int const idx, float const * const b) {
    return GET_COMP(patch, idx, X) * b[0] +
           GET_COMP(patch, idx, Y) * b[1] +
           GET_COMP(patch, idx, Z) * b[2] +
           GET_COMP(patch, idx, W) * b[3];
}

static __global__
void kernelMVP(GpuQueue queue, RenderModel const model) {
    uint64_t patchId = getGlobalIdx3DZ();
    int idx = getIdx();

    if (patchId >= queue.size) {
        return;
    }

    glm::vec4 src;
    glm::vec4 dest;

    uint64_t pointId = patchId * NUM_POINTS + idx;

    src.x = model.x[pointId];
    src.y = model.y[pointId];
    src.z = model.z[pointId];
    src.w = model.w[pointId];

    for (int i = 0; i < 4; i++) {
        dest[i] = cudaDot4D(gMVP[i], src);
    }

    queue.x.getPointer()[pointId] = dest.x;
    queue.y.getPointer()[pointId] = dest.y;
    queue.z.getPointer()[pointId] = dest.z;
    queue.w.getPointer()[pointId] = dest.w;
}


void VDRender::runKernelMVP(GpuQueue &queue, RenderModel const &model) {
    Timer time;
    time.start();

    size_t patches = NUM_PATCHES_MVP;
    dim3 block(DEGREE, DEGREE, patches);
    dim3 gridDim = gridConfigureZ(queue.size, block);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelMVP), hipFuncCachePreferL1);
    kernelMVP <<<gridDim, block>>> (queue, model);
    hipDeviceSynchronize();

    uint64_t elapsed = time.elapsedNanosecondsU64();
    m_statistics.kernelMVPNanoseconds += elapsed;
    cudaCheckErrors("KernelMVP");
}





static __device__ __inline__
void makeEdgesLinear(float volatile *patch, DecisionBits const &decision) {
    int8_t threadShift = threadIdx.x * NUM_POINTS;
    for (int i = threadIdx.y; i < 8; i += 4) {
        float t = ((i & 1) + 1.0f) / 3.0f;
        int8_t point_id = edgeThreadPointChooser[0 * 8 + i];
        int8_t next_point_id = edgeThreadPointChooser[0 * 8 + (i & 1)? i - 1 : i + 1];
        if (decision.get(point_id) && decision.get(next_point_id)) {
            patch[point_id + threadShift] = (1.0f - t) * patch[edgeThreadPointChooser[1 * 8 + i] + threadShift] +
                                      t * patch[edgeThreadPointChooser[2 * 8 + i] + threadShift];
        }
    }
}

static __device__ __inline__
void loadPoints(float volatile *dest1, float volatile *dest2, PatchPointer const &patch, int const idx) {
    GET_COMP(dest1, idx, X) = patch.x[idx];
    GET_COMP(dest1, idx, Y) = patch.y[idx];
    GET_COMP(dest1, idx, Z) = patch.z[idx];
    GET_COMP(dest1, idx, W) = patch.w[idx];

    GET_COMP(dest2, idx, X) = GET_COMP(dest1, idx, X);
    GET_COMP(dest2, idx, Y) = GET_COMP(dest1, idx, Y);
    GET_COMP(dest2, idx, Z) = GET_COMP(dest1, idx, Z);
    GET_COMP(dest2, idx, W) = GET_COMP(dest1, idx, W);
}

static __device__ __inline__
void subdivide4(PatchPointer &out, float volatile *sharedA, float volatile *sharedB) {
    float LL[NUM_COMP];
    float LR[NUM_COMP];
    float RL[NUM_COMP];
    float RR[NUM_COMP];
    int i,j;

    for (i = 0; i < NUM_COMP; i++) {
        LL[i] = 0;
        LR[i] = 0;
        for (j = 0; j < DEGREE; j++) {
            LL[i] += GET_COMP(sharedA, (threadIdx.y * DEGREE + j), i) * gSLU[j][threadIdx.x];
            LR[i] += GET_COMP(sharedA, (threadIdx.y * DEGREE + j), i) * gSRU[j][threadIdx.x];
        }
        GET_COMP(sharedA, (threadIdx.y * DEGREE + threadIdx.x), i) = LL[i];
        GET_COMP(sharedB, (threadIdx.y * DEGREE + threadIdx.x), i) = LR[i];
    }

    for (i = 0; i < NUM_COMP; i++) {
        LL[i] = 0;
        LR[i] = 0;
        RL[i] = 0;
        RR[i] = 0;

        for (j = 0; j < DEGREE; j++) {
            int idx = (j * DEGREE + threadIdx.x) + i * NUM_POINTS;
            LL[i] += gSLV[threadIdx.y][j] * sharedA[idx];
            LR[i] += gSLV[threadIdx.y][j] * sharedB[idx];
            RL[i] += gSRV[threadIdx.y][j] * sharedA[idx];
            RR[i] += gSRV[threadIdx.y][j] * sharedB[idx];
        }
    }


    for (i = 0; i < NUM_COMP; i++) {
        int shift = threadIdx.y * DEGREE + threadIdx.x;
        GET_COMP(out.p[i], shift, X) = LL[i];
        GET_COMP(out.p[i], shift, Y) = LR[i];
        GET_COMP(out.p[i], shift, Z) = RL[i];
        GET_COMP(out.p[i], shift, W) = RR[i];
    }
}

__device__
float sign(float const t) {
    return (t > 0.0f) * 1.0f + (t < 0.0f) * (-1.0f);
}

static __device__
void generatePrimitives(float const factor, float const *patches, Triangles &pointer) {
    int idx = getIdx();
    float *out = reinterpret_cast<float*>(pointer.points);

    int shift = (idx & 1) + 2;
    for (int i = idx / 2; i < NUM_POINTS_IN_TRIANGLES; i += 8) {
        float output = patches[shift * NUM_POINTS + triangleThreadPointChooser[i]];
        out[i * NUM_COMP + shift] = output;
    }

    shift = (idx & 1);
    float maxSize = fmaxf(gWidth, gHeight);
    int halfidx = idx / 2;
    for (int i = halfidx; i < NUM_POINTS_IN_TRIANGLES; i += 8) {
        glm::vec2 a;
        int3 idx;
        idx.x = triangleThreadPointChooser[i];
        idx.y = triangleThreadPointChooser2[i];
        idx.z = triangleThreadPointChooser3[i];

        a.x = patches[0 * NUM_POINTS + idx.x];
        a.y = patches[1 * NUM_POINTS + idx.x];

        float w = patches[3 * NUM_POINTS + idx.x];
        a  /= w;
        a.x = a.x - 0.5f * (patches[0 * NUM_POINTS + idx.y] / patches[3 * NUM_POINTS + idx.y] + patches[0 * NUM_POINTS + idx.z] / patches[3 * NUM_POINTS + idx.z]);
        a.y = a.y - 0.5f * (patches[1 * NUM_POINTS + idx.y] / patches[3 * NUM_POINTS + idx.y] + patches[1 * NUM_POINTS + idx.z] / patches[3 * NUM_POINTS + idx.z]);
        float t = a[shift] * rsqrtf(SQR(a[0]) + SQR(a[1]));
        out[i * NUM_COMP + shift] = patches[shift * NUM_POINTS + idx.x] + 2.0f * (factor / maxSize) * (t) * w;
    }
}

static __device__
void generatePrimitives0(float const *patches, Triangles &pointer) {
    float *out = reinterpret_cast<float*>(pointer.points);
    for (int i = threadIdx.y; i < NUM_POINTS_IN_TRIANGLES; i += NUM_COMP) {
        float output = GET_COMP(patches, triangleThreadPointChooser[i], threadIdx.x);
        out[i * NUM_COMP + threadIdx.x] = output;
    }
}


static __global__
void kernelTriangleGenerate(GpuQueue queue, Triangles *pointer) {
    uint64_t patchId = getGlobalIdx3DZ();
    int idx = getIdx();
    if (patchId >= queue.size) {
        return;
    }

    __shared__ volatile float patches[NUM_PATCHES_WITH_SHARED][NUM_COMP][NUM_POINTS];
    float volatile *shared_pointer = reinterpret_cast<float volatile*>(patches) + threadIdx.z * NUM_COMP * NUM_POINTS;
    GET_COMP(shared_pointer, idx, X) = queue.x.getPointer()[patchId * NUM_POINTS + idx];
    GET_COMP(shared_pointer, idx, Y) = queue.y.getPointer()[patchId * NUM_POINTS + idx];
    GET_COMP(shared_pointer, idx, Z) = queue.z.getPointer()[patchId * NUM_POINTS + idx];
    GET_COMP(shared_pointer, idx, W) = queue.w.getPointer()[patchId * NUM_POINTS + idx];
    generatePrimitives0(const_cast<float*>(shared_pointer), pointer[patchId]);

}
static
void runTriangleGenerate(GpuQueue const &queue, Triangles *pointer) {
    Timer time;
    time.start();

    size_t patches = NUM_PATCHES_WITH_SHARED;
    dim3 block(DEGREE, DEGREE, patches);
    dim3 gridDim = gridConfigureZ(queue.size, block);
    kernelTriangleGenerate <<<gridDim, block>>> (queue, pointer);
    hipDeviceSynchronize();

    Log::getInstance().write(LOG_MESSAGE_TYPE::DEBUG, "ViewDependentRender", "runTriangleGenerate", "total time %f", time.elapsedNanoseconds());
    cudaCheckErrors("runTriangleGenerate");
}


static __device__ __inline__
float signedArea(glm::vec2 const &a, glm::vec2 const &b, glm::vec2 const &c) {
    return (b.x - a.x)*(c.y - a.y) - (b.y - a.y)*(c.x - a.x);
}

static __device__ __inline__
float signedArea(float volatile *patch, int const a, int const b, int const c) {
    return (GET_COMP(patch, b, X) - GET_COMP(patch, a, X))*(GET_COMP(patch, c, Y) - GET_COMP(patch, a, Y))
            - (GET_COMP(patch, b, Y) - GET_COMP(patch, a, Y))*(GET_COMP(patch, c, X) - GET_COMP(patch, a, X));
}

static __device__ __inline__
float signedArea(float const *patch, int const a, int const b, int const c) {
    return (GET_COMP(patch, b, X) - GET_COMP(patch, a, X))*(GET_COMP(patch, c, Y) - GET_COMP(patch, a, Y))
            - (GET_COMP(patch, b, Y) - GET_COMP(patch, a, Y))*(GET_COMP(patch, c, X) - GET_COMP(patch, a, X));
}

__device__
inline float det (float const a, float const b, float const c, float const d) {
    return a * d - b * c;
}

__device__
inline bool between (float const a, float const b, float const c) {
    return fminf(a,b) <= c && c <= fmaxf(a,b);
}

//From http://stackoverflow.com/questions/563198/how-do-you-detect-where-two-line-segments-intersect
__device__
bool insertSegmentOptimized(float volatile *patch, int const a, int const b, int const c, int const d)
{
    float s02_x, s02_y, s10_x, s10_y, s32_x, s32_y, s_numer, t_numer, denom;
    s10_x = GET_COMP(patch, b, X) - GET_COMP(patch, a, X);
    s10_y = GET_COMP(patch, b, Y) - GET_COMP(patch, a, Y);
    s32_x = GET_COMP(patch, d, X) - GET_COMP(patch, c, X);
    s32_y = GET_COMP(patch, d, Y) - GET_COMP(patch, c, Y);

    denom = s10_x * s32_y - s32_x * s10_y;
    if (denom == 0)
        return 0;
    bool denomPositive = denom > 0;

    s02_x = GET_COMP(patch, a, X) - GET_COMP(patch, c,  X);
    s02_y = GET_COMP(patch, a, Y) - GET_COMP(patch, c, Y);
    s_numer = s10_x * s02_y - s10_y * s02_x;
    if ((s_numer < 0) == denomPositive)
        return 0;
    t_numer = s32_x * s02_y - s32_y * s02_x;
    if ((t_numer < 0) == denomPositive)
        return 0;
    if (((s_numer > denom) == denomPositive) || ((t_numer > denom) == denomPositive))
        return 0;
    return true;
}

__device__
void edgeExtend(float factor, float volatile *patch, float volatile *memory) {
    int idx = getIdx();
    GET_COMP(memory, idx, X) = GET_COMP(patch, idx, X) / GET_COMP(patch, idx, W);
    GET_COMP(memory, idx, Y) = GET_COMP(patch, idx, Y) / GET_COMP(patch, idx, W);

    if (idx == 0) {
        bool diagIntersect = (insertSegmentOptimized(memory, 0, 15, 3, 12));
        // проверка, что полигон не имеет самопересечений -- диагонали пересекаются
        if (diagIntersect) {
            glm::vec2 diag1, diag2;
            diag1.x = GET_COMP(memory, 0, X) - GET_COMP(memory, 15, X);
            diag1.y = GET_COMP(memory, 0, Y) - GET_COMP(memory, 15, Y);

            diag2.x = GET_COMP(memory, 3, X) - GET_COMP(memory, 12, X);
            diag2.y = GET_COMP(memory, 3, Y) - GET_COMP(memory, 12, Y);

            diag1 *= rsqrtf(SQR(diag1[0]) + SQR(diag1[1]));
            diag2 *= rsqrtf(SQR(diag2[0]) + SQR(diag2[1]));

            factor = 2.0 * factor / gWidth;
            for (int i = 0; i < 2; i++) {
                GET_COMP(patch, 0, i)  +=  factor * diag1[i] * GET_COMP(patch, 0, W);
                GET_COMP(patch, 3, i)  +=  factor * diag2[i] * GET_COMP(patch, 3, W);
                GET_COMP(patch, 15, i) += factor * -diag1[i] * GET_COMP(patch, 15, W);
                GET_COMP(patch, 12, i) += factor * -diag2[i] * GET_COMP(patch, 12, W);
            }
        }
    }
}

static __global__
void kernelSplit(GpuQueue queue, GpuQueue newQueue, Triangles *primitives,
                 uint64_t const *todoExclusiveSum, uint64_t const *doneExclusiveSum, DecisionBits const *threadDecision)
{
    uint64_t patchId = getGlobalIdx3DZ();
    uint64_t idx = getIdx();

    if (patchId >= queue.size || threadDecision[patchId].isCULL()) {
        return;
    }
    __shared__ volatile float sharedMemory[NUM_PATCHES_SPLIT][2][NUM_COMP][NUM_POINTS];

    float volatile *patchA = reinterpret_cast<float volatile*>(sharedMemory) + threadIdx.z * 2 * NUM_POINTS * NUM_COMP;
    float volatile *patchB = patchA + NUM_POINTS * NUM_COMP;
    PatchPointer patch;
    uint64_t offset = patchId * NUM_POINTS;
    patch.x = queue.x.getPointer() + offset;
    patch.y = queue.y.getPointer() + offset;
    patch.z = queue.z.getPointer() + offset;
    patch.w = queue.w.getPointer() + offset;

    loadPoints(patchA, patchB, patch, idx);

    if (!threadDecision[patchId].isReady()) {
        makeEdgesLinear(patchA, threadDecision[patchId]);

        offset = todoExclusiveSum[patchId] * NUM_POINTS;
        patch.x = newQueue.x.getPointer() + offset;
        patch.y = newQueue.y.getPointer() + offset;
        patch.z = newQueue.z.getPointer() + offset;
        patch.w = newQueue.w.getPointer() + offset;

        subdivide4(patch, patchA, patchB);
    } else {
        //makeEdgesLinear(patchA, threadDecision[patchId]);
        //generatePrimitives(1, const_cast<float*>(patchA), primitives[doneExclusiveSum[patchId]]);
        edgeExtend(0.5, patchA, patchB);
        makeEdgesLinear(patchA, threadDecision[patchId]);
        generatePrimitives0(const_cast<float*>(patchA), primitives[doneExclusiveSum[patchId]]);
    }
}

static __device__ __inline__
bool testBackfaceCulling(glm::vec4* corners, VDFrontFace const &face) {
    glm::vec2 a (corners[0].x / corners[0].w, corners[0].y / corners[0].w);
    glm::vec2 b (corners[1].x / corners[1].w, corners[1].y / corners[1].w);
    glm::vec2 c (corners[2].x / corners[2].w, corners[2].y / corners[2].w);
    glm::vec2 d (corners[3].x / corners[3].w, corners[3].y / corners[3].w);

    if (face == VDFrontFace::FRONT) {
        return (signedArea(a,b,c) > 0 && signedArea (c, b, d) > 0);
    }
    return (signedArea(a,b,c) < 0 && signedArea (c, b, d) < 0);
}


static __device__
bool boundingBox(PatchPointer const &patch, int idx) {
    glm::vec4 point;
    point.x = patch.x[idx];
    point.y = patch.y[idx];
    point.z = patch.z[idx];
    point.w = patch.w[idx];

    uint32_t test;

    for (int i = 0; i < 6; i++) {
        test = ((static_cast<uint32_t>(__ballot(cudaDot4D(point, gClipSpacePlanes[i]) >= 0.0f))) >> ((threadIdx.z & 1)? 16 : 0)) & 0xFFFF;
        if (test == 0) {
            return false;
        }
    }

    return true;
}


static __device__ __inline__
int getCorner(int i) {
    return i * 3 + (i > 1) * 6;
}


static __device__ __inline__
void loadCornersToSharedMemory(float volatile *memory, PatchPointer const &pointer) {
    memory[threadIdx.x * 4 + threadIdx.y] = pointer.p[threadIdx.y][getCorner(threadIdx.x)]; // транспонирование
}



static __device__
DecisionBits approxQuad(PatchPointer const &patch, float const &threshold, glm::vec4 const *corners, int const idx) {
    DecisionBits decision = { 0 };
    glm::vec2 test;
    float w;
    glm::vec4 interpolate = bilinearInterpolation4D(threadIdx.x * 1.0f / 3.0f, threadIdx.y * 1.0f / 3.0f, corners[0], corners[1], corners[2], corners[3]);

    w = patch.w[idx];
    test.x = patch.x[idx] / w;
    test.y = patch.y[idx] / w;

    test.x = (test.x * 0.5f + 0.5f) * gWidth;
    test.y = (test.y * 0.5f + 0.5f) * gHeight;

    interpolate.x = (interpolate.x / interpolate.w * 0.5f + 0.5f) * gWidth;
    interpolate.y = (interpolate.y / interpolate.w * 0.5f + 0.5f) * gHeight;

    bool lessThenTreshold = (SQR((test.x - interpolate.x)) + SQR((test.y - interpolate.y))) <= SQR(threshold);
    decision.arr = ((static_cast<uint32_t>(__ballot(lessThenTreshold))) >> ((threadIdx.z & 1)? 16 : 0)) & 0xFFFF;
    return decision.arr;
}

static __device__ __inline__
glm::vec2 bilinearInterpolationCornersSharedMemory(float const u, float const v, float const *memory) {
    glm::vec2 vec;
    float t1, t2;
    for (int i = 0; i < 2; i++) {
        t1 = linearInterpolation(GET_COMP(memory, 0, i), GET_COMP(memory, 3, i), u);
        t2 = linearInterpolation(GET_COMP(memory, 12, i), GET_COMP(memory, 15, i), u);
        vec[i] = linearInterpolation(t1, t2, v);
    }

    t1 = linearInterpolation(GET_COMP(memory, 0, W), GET_COMP(memory, 3, W), u);
    t2 = linearInterpolation(GET_COMP(memory, 12, W), GET_COMP(memory, 15, W), u);
    float w = linearInterpolation(t1, t2, v);
    vec.x /= w;
    vec.y /= w;
    return vec;
}

static __device__ __inline__
bool testBackfaceCullingSharedMemory(float volatile *memory, VDFrontFace const &face) {
    int idx = getIdx();
    memory[0 * NUM_POINTS + idx] /= memory[3 * NUM_POINTS + idx];
    memory[1 * NUM_POINTS + idx] /= memory[3 * NUM_POINTS + idx];

    if (idx == 0) {
        if (face == VDFrontFace::FRONT) {
            return (signedArea(memory, 0, 3, 12) > 0 && signedArea(memory, 12, 3, 15) > 0);
        }
        return (signedArea(memory, 0, 3, 12) < 0 && signedArea(memory, 12, 3, 15) < 0);
    }
    return false;
}


static __device__
bool boundingBoxSharedMemory(float const *memory, int const idx) {
    uint32_t shift = ((threadIdx.z & 1) * 16);
    for (int i = 0; i < 6; i++) {
        float dot = cudaDot4D(memory, idx, gClipSpacePlanes[i]);
        int test = (((__ballot(dot >= 0.0f))) >> shift) & 0xFFFF;
        if (test == 0) {
            return false;
        }
    }
    return true;
}


static __device__
DecisionBits approxQuadSharedMemory(float *memory, float const &threshold, int const idx) {
    DecisionBits decision = { 0 };
    glm::vec2 test;
    glm::vec2 interpolate = bilinearInterpolationCornersSharedMemory(threadIdx.x * 1.0f / 3.0f, threadIdx.y * 1.0f / 3.0f, memory);

    test.x = GET_COMP(memory, idx, X) / GET_COMP(memory, idx, W);
    test.y = GET_COMP(memory, idx, Y) / GET_COMP(memory, idx, W);

    test.x = (test.x - interpolate.x) * 0.5f * gWidth;
    test.y = (test.y - interpolate.y) * 0.5f * gHeight;

    bool lessThenTreshold = (SQR((test.x)) + SQR((test.y))) <= SQR(threshold);
    decision.arr = ((__ballot(lessThenTreshold)) >> ((threadIdx.z & 1) * 16)) & 0xFFFF;
    return decision.arr;
}

static __device__ __inline__
void loadPointsSharedMemory(float volatile *dest1, PatchPointer const &patch, int const idx) {
    dest1[0 * NUM_POINTS + idx] = patch.x[idx];
    dest1[1 * NUM_POINTS + idx] = patch.y[idx];
    dest1[2 * NUM_POINTS + idx] = patch.z[idx];
    dest1[3 * NUM_POINTS + idx] = patch.w[idx];
}

static __global__
void kernelOracleSharedMemory(GpuQueue const queue, float const threshold,
                  uint64_t *todo, uint64_t *done, DecisionBits *threadDecision, VDFrontFace const face, bool const forceDone)
{
    uint64_t patchId = getGlobalIdx3DZ();
    int idx = getIdx();
    if (patchId >= queue.size) {
        return;
    }

    __shared__ float memory[NUM_PATCHES_ORACLE][NUM_COMP][NUM_POINTS];

    DecisionBits decision = {0};
    PatchPointer patch;

    uint64_t offset = patchId * NUM_POINTS;
    patch.x = queue.x.getPointer() + offset;
    patch.y = queue.y.getPointer() + offset;
    patch.z = queue.z.getPointer() + offset;
    patch.w = queue.w.getPointer() + offset;

    float* halfWarpMemory = reinterpret_cast<float*>(memory) + threadIdx.z * NUM_COMP * NUM_POINTS;

    loadPointsSharedMemory(const_cast<float volatile*>(halfWarpMemory), patch, idx);

    bool inscreen = boundingBoxSharedMemory(halfWarpMemory, idx);

    if (!inscreen) {
        decision.setCULL();
    } else {
        if (forceDone == false) {
            decision = approxQuadSharedMemory(halfWarpMemory, threshold, idx);
        } else {
            decision = 0xFFFF;
        }

        if (decision.isReady() && face != VDFrontFace::NONE) {
            bool backface = testBackfaceCullingSharedMemory(halfWarpMemory, face);
            if (backface) {
                decision.setCULL();
            }
        }
    }

    if (idx == 0) {
        todo[patchId] = (decision.isReady() || decision.isCULL()) ? 0 : 4;
        threadDecision[patchId] = decision;
        done[patchId] = decision.isReady();
    }
}

__global__
void kernelOracle(GpuQueue const queue, float const threshold,
                  uint64_t *todo, uint64_t *done, DecisionBits *threadDecision, VDFrontFace const face, bool const forceDone)
{
    uint64_t patchId = getGlobalIdx3DZ();
    int idx = getIdx();
    if (patchId >= queue.size) {
        return;
    }

    __shared__ float memory[NUM_PATCHES_ORACLE][4][NUM_COMP];
    PatchPointer patch;
    DecisionBits decision = {0};
    patch.x = queue.x.getPointer() + patchId * NUM_POINTS;
    patch.y = queue.y.getPointer() + patchId * NUM_POINTS;
    patch.z = queue.z.getPointer() + patchId * NUM_POINTS;
    patch.w = queue.w.getPointer() + patchId * NUM_POINTS;

    float* halfWarpMemory = reinterpret_cast<float*>(memory) + threadIdx.z * 4 * NUM_COMP;
    bool inscreen = boundingBox(patch, idx);

    if (!inscreen) {
        decision.setCULL();
    } else {
        // считываем угловые точки
        loadCornersToSharedMemory(halfWarpMemory, patch);

        if (forceDone == false) {
            decision = approxQuad(patch, threshold, reinterpret_cast<glm::vec4*>(halfWarpMemory), idx);
        } else {
            decision = 0xFFFF;
        }

        if (idx == 0 && decision.isReady() && face != VDFrontFace::NONE) {
            bool backface = testBackfaceCulling(reinterpret_cast<glm::vec4*>(halfWarpMemory), face);
            if (backface) {
                decision.setCULL();
            }
        }
    }

    if (idx == 0) {
        todo[patchId] = (decision.isReady() || decision.isCULL()) ? 0 : 4;
        threadDecision[patchId] = decision;
        done[patchId] = decision.isReady();
    }
}


void VDRender::runKernelOracle(GpuQueue &queue,
                               float threshold,
                               GpuPointer<uint64_t> &todo,
                               GpuPointer<uint64_t> &done,
                               GpuPointer<DecisionBits> &threadDecision,
                               VDFrontFace const &face,
                               bool const forceDone) {
    size_t patches = NUM_PATCHES_ORACLE;
    dim3 block = dim3(DEGREE, DEGREE, patches);
    dim3 gridDim = gridConfigureZ(queue.size, block);
    Timer time;
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelOracleSharedMemory), hipFuncCachePreferShared);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelOracle), hipFuncCachePreferShared);

    time.start();
    kernelOracleSharedMemory<<<gridDim, block>>>(queue,
                                     threshold,
                                     todo.getPointer(),
                                     done.getPointer(),
                                     threadDecision.getPointer(),
                                     face,
                                     forceDone);

    hipDeviceSynchronize();
    uint64_t elapsed = time.elapsedNanosecondsU64();
    m_statistics.kernelOracleNanoseconds += elapsed;
    cudaCheckErrors("KernelOracle");
}


bool VDRender::runKernelScan(GpuPointer<uint64_t> const &array, uint64_t const size, uint64_t &sum, GpuPointer<uint64_t> &exclusiveSum) {


    size_t cub_tmp_memory_size = 0;
    hipcub::DeviceScan::ExclusiveSum(nullptr, cub_tmp_memory_size, array.getPointer(), exclusiveSum.getPointer(), size);

    GpuPointer<char> cub_tmp_memory = SystemManager::getInstance()->gpuStackAllocator.alloc<char>(cub_tmp_memory_size);

    if (cub_tmp_memory.getPointer() == nullptr) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::DEBUG, "ViewDependentRender", "runKernelScan", "Allocation problem %s", SourcePos().c_str());
        return false;
    }

    Timer time;
    time.start();
    hipcub::DeviceScan::ExclusiveSum(cub_tmp_memory.getPointer(), cub_tmp_memory_size, array.getPointer(), exclusiveSum.getPointer(), size);
    hipDeviceSynchronize();
    uint64_t elapsed = time.elapsedNanosecondsU64();
    m_statistics.kernelScanNanoseconds += elapsed;

    sum = 0;
    hipMemcpy(&sum, exclusiveSum.getPointer() + (size - 1), sizeof(uint64_t), hipMemcpyDeviceToHost);
    SystemManager::getInstance()->gpuStackAllocator.free(cub_tmp_memory);
    cudaCheckErrors("KernelScan");
    return true;
}

void VDRender::runKernelSplit(GpuQueue &queue,
                                     GpuQueue &newQueue,
                                     Triangles *triangles,
                                     GpuPointer<uint64_t> todoExclusiveSum,
                                     GpuPointer<uint64_t> doneExclusiveSum,
                                     GpuPointer<DecisionBits> threadDecisionBits) {
    size_t patches = NUM_PATCHES_SPLIT;
    dim3 block = dim3(4, 4, patches);
    dim3 gridDim = gridConfigureZ(queue.size, block);
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(kernelSplit), hipFuncCachePreferShared);
    Timer time;
    time.start();
    kernelSplit<<<gridDim, block>>>(queue,
                                    newQueue,
                                    triangles,
                                    todoExclusiveSum.getPointer(),
                                    doneExclusiveSum.getPointer(),
                                    threadDecisionBits.getPointer());
    hipDeviceSynchronize();
    uint64_t elapsed = time.elapsedNanosecondsU64();
    m_statistics.kernelSplitNanoseconds += elapsed;
    cudaCheckErrors("KernelSplit");
}

void VDRender::flushGL() {
    if (m_glPart.objectsInVBO > 0) {
        drawGL(m_glPart.objectsInVBO, 1);
        m_statistics.maxMemoryUsedGLBufferInMB = std::max(CONVERT_BYTES_TO_MB(m_glPart.objectsInVBO * sizeof(Triangles)), m_statistics.maxMemoryUsedGLBufferInMB);
        m_glPart.objectsInVBO = 0;
    }
}

void VDRender::render(std::string const name, float threshold, int maxlevel) {
    Timer time;
    time.start();
    auto model_it = this->m_models.find(name);
    if (model_it == m_models.end()) {
        return;
    }
    RenderModel const &model = (*model_it).second;

    if (model.size >= m_settings.maxQueueSize) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::ERROR, "ViewDependentRender", "render", "model.size >= m_settings.maxQueueSize! %zu>=%zu!", model.size, m_settings.maxQueueSize);
        return;
    }
    GpuStackAllocator &allocator = SystemManager::getInstance()->gpuStackAllocator;
    GpuQueue queue[2];
    int first = 0;
    int second = 1;
    int level = 1;

    allocator.pushPosition();
    queue[0].x = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[0].y = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[0].z = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[0].w = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[1].x = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[1].y = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[1].z = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);
    queue[1].w = allocator.alloc<float>(m_settings.maxQueueSize * NUM_POINTS);

    GpuPointer<uint64_t> todo = allocator.alloc<uint64_t>(m_settings.maxQueueSize + 1);
    GpuPointer<uint64_t> done = allocator.alloc<uint64_t>(m_settings.maxQueueSize + 1);
    GpuPointer<uint64_t> todoExclusiveSum = allocator.alloc<uint64_t>(m_settings.maxQueueSize + 1);
    GpuPointer<uint64_t> doneExclusiveSum = allocator.alloc<uint64_t>(m_settings.maxQueueSize + 1);
    GpuPointer<DecisionBits> threadDecision = allocator.alloc<DecisionBits>(m_settings.maxQueueSize + 1);

    if (!queue[0].x.getPointer() || !queue[0].y.getPointer() || !queue[0].z.getPointer() || !queue[0].w.getPointer() ||
        !queue[1].x.getPointer() || !queue[1].y.getPointer() || !queue[1].z.getPointer() || !queue[1].w.getPointer() ||
        !todo.getPointer() || !done.getPointer() || !threadDecision.getPointer()) {
        Log::getInstance().write(LOG_MESSAGE_TYPE::WARNING, "ViewDependentRender", "render", "Allocation problem %s", SourcePos().c_str());
        allocator.popPosition();
        return;
    }

    queue[0].size = model.size;
    queue[1].size = model.size;
    m_glPart.buffer.mapResource();
    uint64_t maxQueueSize = 0;
    uint64_t todoSize = 0;
    uint64_t doneSize = 0;
    runKernelMVP(queue[first], model);

    bool forceDone = false;

    while (queue[first].size > 0) {
        m_statistics.patchesCountTotalProcessed += queue[first].size;
        maxQueueSize = std::max(maxQueueSize, queue[first].size);
        runKernelOracle(queue[first], threshold, todo, done, threadDecision, m_settings.faceMode, forceDone);
        todoSize = 0;
        doneSize = 0;

        if (!runKernelScan(todo, queue[first].size + 1, todoSize, todoExclusiveSum)) {
            Log::getInstance().write(LOG_MESSAGE_TYPE::WARNING, "ViewDependentRender", "render", "runKernelScan failed; break %s", SourcePos().c_str());
            break;
        }

        if (!runKernelScan(done, queue[first].size + 1, doneSize, doneExclusiveSum)) {
            Log::getInstance().write(LOG_MESSAGE_TYPE::WARNING, "ViewDependentRender", "render", "runKernelScan failed; break %s", SourcePos().c_str());
            break;
        }

        if ((todoSize >= m_settings.maxQueueSize || level == maxlevel) && forceDone == false) {
            Log::getInstance().write(LOG_MESSAGE_TYPE::WARNING, "ViewDependentRender", "render", "break %s", SourcePos().c_str());
            forceDone = true;
            continue;
        }


        size_t tmp;
        Triangles *primitivesPtr = reinterpret_cast<Triangles*>(m_glPart.buffer.mappedPointer(tmp));

        if ((m_glPart.objectsInVBO + doneSize) <= m_settings.maxQueueSize) {
            primitivesPtr += m_glPart.objectsInVBO;
            m_glPart.objectsInVBO += doneSize;
        } else {
            m_glPart.buffer.unmapResource();
            flushGL();
            m_glPart.buffer.mapResource();
            m_glPart.objectsInVBO = doneSize;
        }

        runKernelSplit(queue[first], queue[second], primitivesPtr, todoExclusiveSum, doneExclusiveSum, threadDecision);
//        m_glPart.buffer.unmapResource();
//        drawGL(doneSize, level);
//        m_glPart.buffer.mapResource();
        std::swap(first, second);
        queue[first].size = todoSize;
        level++;
    }

    m_statistics.maxMemoryUsedQueueMB = std::max(CONVERT_BYTES_TO_MB(maxQueueSize * sizeof(float) * NUM_POINTS * NUM_COMP * 2), m_statistics.maxMemoryUsedQueueMB);
    m_glPart.buffer.unmapResource();
    allocator.popPosition();
    m_statistics.total += time.elapsedNanosecondsU64();
}

