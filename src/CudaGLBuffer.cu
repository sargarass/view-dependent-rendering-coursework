#include "CudaGLBuffer.cuh"

CudaGLBuffer::CudaGLBuffer() {
    resource = nullptr;
    buffer = 0;
}

CudaGLBuffer::CudaGLBuffer (GLuint buf, unsigned int flags) {
    init(buf, flags);
}

void CudaGLBuffer::init(GLuint buf, unsigned int flags) {
    buffer = buf;
    glBindBuffer(GL_ARRAY_BUFFER, buffer);
    hipGraphicsGLRegisterBuffer ( &resource, buffer, flags );
    glBindBuffer(GL_ARRAY_BUFFER, 0);

}

void CudaGLBuffer::deinit() {
    if (resource != nullptr) {
        hipGraphicsUnregisterResource(resource);
        resource = nullptr;
        buffer = 0;
    }
}

CudaGLBuffer::~CudaGLBuffer() {
    deinit();
}

bool CudaGLBuffer::mapResource() {
    return hipGraphicsMapResources (1, &resource) == hipSuccess;
}

bool CudaGLBuffer::unmapResource() {
    return hipGraphicsUnmapResources (1, &resource) == hipSuccess;
}

void *CudaGLBuffer::mappedPointer(size_t& numBytes) const {
    void *ptr;
    if (hipGraphicsResourceGetMappedPointer (&ptr, &numBytes, resource) != hipSuccess)
        return NULL;
    return ptr;
}

GLuint CudaGLBuffer::getId() const {
    return buffer;
}

hipGraphicsResource *CudaGLBuffer::getResource() const {
    return resource;
}
